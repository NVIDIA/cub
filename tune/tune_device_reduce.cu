#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Evaluates different tuning configurations of DeviceReduce.
 *
 * The best way to use this program:
 * (1) Find the best all-around single-block tune for a given arch.
 *     For example, 1000 samples [1 ..512], 10 timing iterations per config per sample:
 *         ./bin/tune_device_reduce_sm200_nvvm_5.0_abi_i386 --i=10 --s=1000 --n=512 --single --device=0
 * (2) Update the single tune in device_reduce.cuh
 * (3) Find the best all-around multi-block tune for a given arch.
 *     For example, 1000 samples [single-block tile-size ..  50,331,648], 2 timing iterations per config per sample:
 *         ./bin/tune_device_reduce_sm200_nvvm_5.0_abi_i386 --i=2 --s=1000 --device=0
 * (4) Update the multi-block tune in device_reduce.cuh
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <vector>
#include <algorithm>
#include <stdio.h>
#include <hipcub/hipcub.hpp>
#include "../test/test_util.h"

using namespace hipcub;
using namespace std;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

#ifndef TUNE_ARCH
#define TUNE_ARCH 100
#endif

int     g_max_items         = 48 * 1024 * 1024;
int     g_samples           = 100;
int     g_iterations        = 2;
bool    g_verbose           = false;
bool    g_single            = false;
bool    g_verify            = true;


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize problem
 */
template <typename T>
void Initialize(
    int             gen_mode,
    T               *h_in,
    int             num_items)
{
    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
    }
}

/**
 * Sequential reduction
 */
template <typename T, typename ReductionOp>
T Reduce(
    T               *h_in,
    ReductionOp     reduction_op,
    int             num_items)
{
    T retval = h_in[0];
    for (int i = 1; i < num_items; ++i)
        retval = reduction_op(retval, h_in[i]);
    return retval;
}



//---------------------------------------------------------------------
// Full tile test generation
//---------------------------------------------------------------------



/**
 * Wrapper structure for generating and running different tuning configurations
 */
template <
    typename T,
    typename SizeT,
    typename ReductionOp>
struct Schmoo
{
    //---------------------------------------------------------------------
    // Types
    //---------------------------------------------------------------------

    /// Pairing of kernel function pointer and corresponding dispatch params
    template <typename KernelPtr>
    struct DispatchTuple
    {
        KernelPtr                           kernel_ptr;
        DeviceReduce::KernelDispachParams   params;

        float                               avg_throughput;
        float                               best_avg_throughput;
        SizeT                               best_size;
        float                               hmean_speedup;


        DispatchTuple() :
            kernel_ptr(0),
            avg_throughput(0.0),
            best_avg_throughput(0.0),
            hmean_speedup(0.0),
            best_size(0) {}
    };

    /**
     * Comparison operator for DispatchTuple.avg_throughput
     */
    template <typename Tuple>
    static bool MinSpeedup(const Tuple &a, const Tuple &b)
    {
        float delta = a.hmean_speedup - b.hmean_speedup;

        return ((delta < 0.005) && (delta > -0.005)) ?
            (a.best_avg_throughput < b.best_avg_throughput) :       // Negligible average performance differences: defer to best performance
            (a.hmean_speedup < b.hmean_speedup);
    }



    /// Multi-block reduction kernel type and dispatch tuple type
    typedef void (*MultiBlockDeviceReduceKernelPtr)(T*, T*, SizeT, GridEvenShare<SizeT>, GridQueue<SizeT>, ReductionOp);
    typedef DispatchTuple<MultiBlockDeviceReduceKernelPtr> MultiDispatchTuple;

    /// Single-block reduction kernel type and dispatch tuple type
    typedef void (*SingleBlockDeviceReduceKernelPtr)(T*, T*, SizeT, ReductionOp);
    typedef DispatchTuple<SingleBlockDeviceReduceKernelPtr> SingleDispatchTuple;


    //---------------------------------------------------------------------
    // Fields
    //---------------------------------------------------------------------

    vector<MultiDispatchTuple> multi_kernels;       // List of generated multi-block kernels
    vector<SingleDispatchTuple> single_kernels;     // List of generated single-block kernels


    //---------------------------------------------------------------------
    // Kernel enumeration methods
    //---------------------------------------------------------------------

    /**
     * Must have smem that fits in the SM
     * Must have vector load length that divides items per thread
     */
    template <typename BlockReduceTilesPolicy>
    struct SmemSize
    {
        enum
        {
            BYTES = sizeof(typename BlockReduceTiles<BlockReduceTilesPolicy, T*, SizeT>::SmemStorage),
            IS_OK = ((BYTES < ArchProps<TUNE_ARCH>::SMEM_BYTES) &&
                     (BlockReduceTilesPolicy::ITEMS_PER_THREAD % BlockReduceTilesPolicy::VECTOR_LOAD_LENGTH == 0))
        };
    };


    /**
     * Specialization that allows kernel generation with the specified BlockReduceTilesPolicy
     */
    template <
        typename BlockReduceTilesPolicy,
        bool IsOk = SmemSize<BlockReduceTilesPolicy>::IS_OK>
    struct Ok
    {
        /// Enumerate multi-block kernel and add to the list
        template <typename KernelsVector>
        static void GenerateMulti(
            KernelsVector &multi_kernels,
            int subscription_factor)
        {
            MultiDispatchTuple tuple;
            tuple.params.template Init<BlockReduceTilesPolicy>(subscription_factor);
            tuple.kernel_ptr = MultiBlockDeviceReduceKernel<BlockReduceTilesPolicy, T*, T*, SizeT, ReductionOp>;
            multi_kernels.push_back(tuple);
        }


        /// Enumerate single-block kernel and add to the list
        template <typename KernelsVector>
        static void GenerateSingle(KernelsVector &single_kernels)
        {
            SingleDispatchTuple tuple;
            tuple.params.template Init<BlockReduceTilesPolicy>();
            tuple.kernel_ptr = SingleBlockDeviceReduceKernel<BlockReduceTilesPolicy, T*, T*, SizeT, ReductionOp>;
            single_kernels.push_back(tuple);
        }
    };

    /**
     * Specialization that rejects kernel generation with the specified BlockReduceTilesPolicy
     */
    template <typename BlockReduceTilesPolicy>
    struct Ok<BlockReduceTilesPolicy, false>
    {
        template <typename KernelsVector>
        static void GenerateMulti(KernelsVector &multi_kernels, int subscription_factor) {}

        template <typename KernelsVector>
        static void GenerateSingle(KernelsVector &single_kernels) {}
    };


    /// Enumerate block-scheduling variations
    template <
        int                     BLOCK_THREADS,
        int                     ITEMS_PER_THREAD,
        int                     VECTOR_LOAD_LENGTH,
        BlockReduceAlgorithm    BLOCK_ALGORITHM,
        PtxLoadModifier         LOAD_MODIFIER>
    void Enumerate()
    {
        // Multi-block kernels
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_EVEN_SHARE> >::GenerateMulti(multi_kernels, 1);
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_EVEN_SHARE> >::GenerateMulti(multi_kernels, 2);
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_EVEN_SHARE> >::GenerateMulti(multi_kernels, 4);
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_EVEN_SHARE> >::GenerateMulti(multi_kernels, 8);
#if TUNE_ARCH >= 200
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_DYNAMIC> >::GenerateMulti(multi_kernels, 1);
#endif

        // Single-block kernels
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_EVEN_SHARE> >::GenerateSingle(single_kernels);
    }


    /// Enumerate load modifier variations
    template <
        int                     BLOCK_THREADS,
        int                     ITEMS_PER_THREAD,
        int                     VECTOR_LOAD_LENGTH,
        BlockReduceAlgorithm    BLOCK_ALGORITHM>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, PTX_LOAD_NONE>();
#if TUNE_ARCH >= 350
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, PTX_LOAD_LDG>();
#endif
    }


    /// Enumerate block algorithms
    template <
        int BLOCK_THREADS,
        int ITEMS_PER_THREAD,
        int VECTOR_LOAD_LENGTH>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_REDUCE_RAKING>();
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_REDUCE_WARP_REDUCTIONS>();
    }


    /// Enumerate vectorization variations
    template <
        int BLOCK_THREADS,
        int ITEMS_PER_THREAD>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, 1>();
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, 2>();
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, 4>();
    }


    /// Enumerate thread-granularity variations
    template <int BLOCK_THREADS>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, 1>();
        Enumerate<BLOCK_THREADS, 2>();
        Enumerate<BLOCK_THREADS, 4>();
//      Enumerate<BLOCK_THREADS, 7>();
        Enumerate<BLOCK_THREADS, 8>();
        Enumerate<BLOCK_THREADS, 9>();
        Enumerate<BLOCK_THREADS, 11>();
        Enumerate<BLOCK_THREADS, 12>();
//      Enumerate<BLOCK_THREADS, 13>();
//      Enumerate<BLOCK_THREADS, 15>();
        Enumerate<BLOCK_THREADS, 16>();
//      Enumerate<BLOCK_THREADS, 17>();
//      Enumerate<BLOCK_THREADS, 19>();
        Enumerate<BLOCK_THREADS, 20>();
//      Enumerate<BLOCK_THREADS, 21>();
//      Enumerate<BLOCK_THREADS, 23>();
        Enumerate<BLOCK_THREADS, 24>();
//      Enumerate<BLOCK_THREADS, 25>();
    }


    /// Enumerate block size variations
    void Enumerate()
    {
        printf("\nEnumerating kernels\n"); fflush(stdout);

        Enumerate<32>();
        Enumerate<64>();
        Enumerate<96>();
        Enumerate<128>();
        Enumerate<160>();
        Enumerate<192>();
        Enumerate<256>();
        Enumerate<512>();
    }


    //---------------------------------------------------------------------
    // Multi-block test methods
    //---------------------------------------------------------------------

    /**
     * Test multi reduction
     */
    void Test(
        MultiDispatchTuple      &multi_dispatch,
        SingleDispatchTuple     &single_dispatch,
        T*                      d_in,
        T*                      d_out,
        T*                      h_reference,
        SizeT                   num_items,
        ReductionOp             reduction_op)
    {
        // Clear output
        if (g_verify) CubDebugExit(hipMemset(d_out, 0, sizeof(T)));

        // Warmup/correctness iteration
        DeviceReduce::Dispatch(
            multi_dispatch.kernel_ptr,
            single_dispatch.kernel_ptr,
            multi_dispatch.params,
            single_dispatch.params,
            d_in,
            d_out,
            num_items,
            reduction_op);

        if (g_verify) CubDebugExit(hipDeviceSynchronize());

        // Copy out and display results
        int compare = (g_verify) ?
            CompareDeviceResults(h_reference, d_out, 1, true, false) :
            0;

        // Performance
        GpuTimer gpu_timer;
        float elapsed_millis = 0.0;
        for (int i = 0; i < g_iterations; i++)
        {
            gpu_timer.Start();

            DeviceReduce::Dispatch(
                multi_dispatch.kernel_ptr,
                single_dispatch.kernel_ptr,
                multi_dispatch.params,
                single_dispatch.params,
                d_in,
                d_out,
                num_items,
                reduction_op);

            gpu_timer.Stop();
            elapsed_millis += gpu_timer.ElapsedMillis();
        }

        float avg_elapsed = elapsed_millis / g_iterations;
        float avg_throughput = float(num_items) / avg_elapsed / 1000.0 / 1000.0;
        float avg_bandwidth = avg_throughput * sizeof(T);

        multi_dispatch.avg_throughput = CUB_MAX(avg_throughput, multi_dispatch.avg_throughput);
        if (avg_throughput > multi_dispatch.best_avg_throughput)
        {
            multi_dispatch.best_avg_throughput = avg_throughput;
            multi_dispatch.best_size = num_items;
        }

        single_dispatch.avg_throughput = CUB_MAX(avg_throughput, single_dispatch.avg_throughput);
        if (avg_throughput > single_dispatch.best_avg_throughput)
        {
            single_dispatch.best_avg_throughput = avg_throughput;
            single_dispatch.best_size = num_items;
        }

        if (g_verbose)
        {
            printf("\t%.2f GB/s, multi_dispatch( ", avg_bandwidth);
            multi_dispatch.params.Print();
            printf(" ), single_dispatch( ");
            single_dispatch.params.Print();
            printf(" )\n");
            fflush(stdout);
        }

        AssertEquals(0, compare);
    }


    /**
     * Evaluate multi-block configurations
     */
    void TestMulti(
        T*                      h_in,
        T*                      d_in,
        T*                      d_out,
        ReductionOp             reduction_op)
    {
        // Simple single kernel tuple for use with multi kernel sweep
        typedef typename DeviceReduce::TunedPolicies<T, SizeT, TUNE_ARCH>::SinglePolicy SimpleSinglePolicy;
        SingleDispatchTuple simple_single_tuple;
        simple_single_tuple.params.template Init<SimpleSinglePolicy>();
        simple_single_tuple.kernel_ptr = SingleBlockDeviceReduceKernel<SimpleSinglePolicy, T*, T*, SizeT, ReductionOp>;

        double max_exponent      = log2(double(g_max_items));
        double min_exponent      = log2(double(simple_single_tuple.params.tile_size));
        unsigned int max_int     = (unsigned int) -1;

        for (int sample = 0; sample < g_samples; ++sample)
        {
            printf("\nMulti-block sample %d, ", sample);

            int num_items;
            if (sample == 0)
            {
                // First sample: use max items
                num_items = g_max_items;
                printf("num_items: %d", num_items); fflush(stdout);
            }
            else
            {
                // Sample a problem size from [2^g_min_exponent, g_max_items].  First 2/3 of the samples are log-distributed, the other 1/3 are uniformly-distributed.
                unsigned int bits;
                RandomBits(bits);
                double scale = double(bits) / max_int;

                if (sample < (2 * g_samples) / 3)
                {
                    // log bias
                    double exponent = ((max_exponent - min_exponent) * scale) + min_exponent;
                    num_items = pow(2.0, exponent);
                    printf("num_items: %d (2^%.2f)", num_items, exponent); fflush(stdout);
                }
                else
                {
                    // uniform bias
                    num_items = CUB_MAX(pow(2.0, min_exponent), scale * g_max_items);
                    printf("num_items: %d (%.2f * %d)", num_items, scale, g_max_items); fflush(stdout);
                }
            }
            if (g_verbose)
                printf("\n");
            else
                printf(", ");

            // Compute reference
            T h_reference = Reduce(h_in, reduction_op, num_items);

            // Run test on each multi-kernel configuration
            float best_avg_throughput = 0.0;
            for (int j = 0; j < multi_kernels.size(); ++j)
            {
                multi_kernels[j].avg_throughput = 0.0;

                Test(multi_kernels[j], simple_single_tuple, d_in, d_out, &h_reference, num_items, reduction_op);

                best_avg_throughput = CUB_MAX(best_avg_throughput, multi_kernels[j].avg_throughput);
            }

            // Print best throughput for this problem size
            printf("Best: %.2fe9 items/s (%.2f GB/s)\n", best_avg_throughput, best_avg_throughput * sizeof(T));

            // Accumulate speedup (inverse for harmonic mean)
            for (int j = 0; j < multi_kernels.size(); ++j)
                multi_kernels[j].hmean_speedup += best_avg_throughput / multi_kernels[j].avg_throughput;
        }

        // Find max overall throughput and compute hmean speedups
        float overall_max_throughput = 0.0;
        for (int j = 0; j < multi_kernels.size(); ++j)
        {
            overall_max_throughput = CUB_MAX(overall_max_throughput, multi_kernels[j].best_avg_throughput);
            multi_kernels[j].hmean_speedup = float(g_samples) / multi_kernels[j].hmean_speedup;
        }

        // Sort by cumulative speedup
        sort(multi_kernels.begin(), multi_kernels.end(), MinSpeedup<MultiDispatchTuple>);

        // Print ranked multi configurations
        printf("\nRanked multi_kernels:\n");
        for (int j = 0; j < multi_kernels.size(); ++j)
        {
            printf("\t (%d) params( ", multi_kernels.size() - j);
            multi_kernels[j].params.Print();
            printf(" ) hmean speedup: %.3f, best throughput %.2f @ %d elements (%.2f GB/s, %.2f%%)\n",
                multi_kernels[j].hmean_speedup,
                multi_kernels[j].best_avg_throughput,
                (int) multi_kernels[j].best_size,
                multi_kernels[j].best_avg_throughput * sizeof(T),
                multi_kernels[j].best_avg_throughput / overall_max_throughput);
        }

        printf("\nMax multi-block throughput %.2f (%.2f GB/s)\n", overall_max_throughput, overall_max_throughput * sizeof(T));
    }


    //---------------------------------------------------------------------
    // Single-block test methods
    //---------------------------------------------------------------------

    /**
     * Test single reduction
     */
    void Test(
        SingleDispatchTuple     &single_dispatch,
        T*                      d_in,
        T*                      d_out,
        T*                      h_reference,
        SizeT                   num_items,
        ReductionOp             reduction_op)
    {
        // Clear output
        if (g_verify) CubDebugExit(hipMemset(d_out, 0, sizeof(T)));

        // Warmup/correctness iteration
        DeviceReduce::DispatchSingle(
            single_dispatch.kernel_ptr,
            single_dispatch.params,
            d_in,
            d_out,
            num_items,
            reduction_op);

        if (g_verify) CubDebugExit(hipDeviceSynchronize());

        // Copy out and display results
        int compare = (g_verify) ?
            CompareDeviceResults(h_reference, d_out, 1, true, false) :
            0;

        // Performance
        GpuTimer gpu_timer;
        float elapsed_millis = 0.0;
        for (int i = 0; i < g_iterations; i++)
        {
            gpu_timer.Start();

            DeviceReduce::DispatchSingle(
                single_dispatch.kernel_ptr,
                single_dispatch.params,
                d_in,
                d_out,
                num_items,
                reduction_op);

            gpu_timer.Stop();
            elapsed_millis += gpu_timer.ElapsedMillis();
        }

        float avg_elapsed = elapsed_millis / g_iterations;
        float avg_throughput = float(num_items) / avg_elapsed / 1000.0 / 1000.0;
        float avg_bandwidth = avg_throughput * sizeof(T);

        single_dispatch.avg_throughput = CUB_MAX(avg_throughput, single_dispatch.avg_throughput);
        if (avg_throughput > single_dispatch.best_avg_throughput)
        {
            single_dispatch.best_avg_throughput = avg_throughput;
            single_dispatch.best_size = num_items;
        }

        if (g_verbose)
        {
            printf("\t%.2f GB/s, single_dispatch( ", avg_bandwidth);
            single_dispatch.params.Print();
            printf(" )\n");
            fflush(stdout);
        }

        AssertEquals(0, compare);
    }


    /**
     * Evaluate single-block configurations
     */
    void TestSingle(
        T*                      h_in,
        T*                      d_in,
        T*                      d_out,
        ReductionOp             reduction_op)
     {
        double max_exponent     = log2(double(g_max_items));
        unsigned int max_int    = (unsigned int) -1;

        for (int sample = 0; sample < g_samples; ++sample)
        {
            printf("\nSingle-block sample %d, ", sample);

            int num_items;
            if (sample == 0)
            {
                // First sample: use max items
                num_items = g_max_items;
                printf("num_items: %d", num_items); fflush(stdout);
            }
            else
            {
                // Sample a problem size from [2, g_max_items], log-distributed
                unsigned int bits;
                RandomBits(bits);
                double scale = double(bits) / max_int;
                double exponent = ((max_exponent - 1) * scale) + 1;
                num_items = pow(2.0, exponent);
                printf("num_items: %d (2^%.2f)", num_items, exponent); fflush(stdout);
            }

            if (g_verbose)
                printf("\n");
            else
                printf(", ");

            // Compute reference
            T h_reference = Reduce(h_in, reduction_op, num_items);

            // Run test on each single-kernel configuration
            float best_avg_throughput = 0.0;
            for (int j = 0; j < single_kernels.size(); ++j)
            {
                single_kernels[j].avg_throughput = 0.0;

                Test(single_kernels[j], d_in, d_out, &h_reference, num_items, reduction_op);

                best_avg_throughput = CUB_MAX(best_avg_throughput, single_kernels[j].avg_throughput);
            }

            // Print best throughput for this problem size
            printf("Best: %.2fe9 items/s (%.2f GB/s)\n", best_avg_throughput, best_avg_throughput * sizeof(T));

            // Accumulate speedup (inverse for harmonic mean)
            for (int j = 0; j < single_kernels.size(); ++j)
                single_kernels[j].hmean_speedup += best_avg_throughput / single_kernels[j].avg_throughput;
        }

        // Find max overall throughput and compute hmean speedups
        float overall_max_throughput = 0.0;
        for (int j = 0; j < single_kernels.size(); ++j)
        {
            overall_max_throughput = CUB_MAX(overall_max_throughput, single_kernels[j].best_avg_throughput);
            single_kernels[j].hmean_speedup = float(g_samples) / single_kernels[j].hmean_speedup;
        }

        // Sort by cumulative speedup
        sort(single_kernels.begin(), single_kernels.end(), MinSpeedup<SingleDispatchTuple>);

        // Print ranked single configurations
        printf("\nRanked single_kernels:\n");
        for (int j = 0; j < single_kernels.size(); ++j)
        {
            printf("\t (%d) params( ", single_kernels.size() - j);
            single_kernels[j].params.Print();
            printf(" ) hmean speedup: %.3f, best throughput %.2f @ %d elements (%.2f GB/s, %.2f%%)\n",
                single_kernels[j].hmean_speedup,
                single_kernels[j].best_avg_throughput,
                (int) single_kernels[j].best_size,
                single_kernels[j].best_avg_throughput * sizeof(T),
                single_kernels[j].best_avg_throughput / overall_max_throughput);
        }

        printf("\nMax single-block throughput %.2f (%.2f GB/s)\n", overall_max_throughput, overall_max_throughput * sizeof(T));
    }

};



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    args.GetCmdLineArgument("n", g_max_items);
    args.GetCmdLineArgument("s", g_samples);
    args.GetCmdLineArgument("i", g_iterations);
    g_verbose = args.CheckCmdLineFlag("v");
    g_single = args.CheckCmdLineFlag("single");
    g_verify = !args.CheckCmdLineFlag("noverify");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--n=<max items>]"
            "[--s=<samples>]"
            "[--i=<timing iterations>]"
            "[--single]"
            "[--v]"
            "[--noverify]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

#if (TUNE_SIZE == 1)
    typedef unsigned char T;
#elif (TUNE_SIZE == 2)
    typedef unsigned short T;
#elif (TUNE_SIZE == 4)
    typedef unsigned int T;
#elif (TUNE_SIZE == 8)
    typedef unsigned long long T;
#else
    // Default
    typedef unsigned int T;
#endif

    typedef unsigned int SizeT;
    Sum<T> reduction_op;

    // Enumerate kernels
    Schmoo<T, SizeT, Sum<T> > schmoo;
    schmoo.Enumerate();

    // Allocate host arrays
    T *h_in = new T[g_max_items];

    // Initialize problem
    Initialize(UNIFORM, h_in, g_max_items);

    // Initialize device arrays
    T *d_in = NULL;
    T *d_out = NULL;
    CubDebugExit(DeviceAllocate((void**)&d_in, sizeof(T) * g_max_items));
    CubDebugExit(DeviceAllocate((void**)&d_out, sizeof(T) * 1));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * g_max_items, hipMemcpyHostToDevice));

    // Test kernels
    if (g_single)
        schmoo.TestSingle(h_in, d_in, d_out, reduction_op);
    else
        schmoo.TestMulti(h_in, d_in, d_out, reduction_op);

    // Cleanup
    if (h_in) delete[] h_in;
    if (d_in) CubDebugExit(DeviceFree(d_in));
    if (d_out) CubDebugExit(DeviceFree(d_out));

    return 0;
}



