#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple demonstration of hipcub::BlockSort
 *
 * Example compilation string:
 *
 * nvcc example_block_radix_sort.cu -gencode=arch=compute_20,code=\"sm_20,compute_20\" -o example_block_radix_sort -m32 -Xptxas -v -I../cub
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console (define before including cub.h)
#define HIPCUB_STDERR

#if defined(_WIN32) || defined(_WIN64)
    #include <windows.h>
    #undef small            // Windows is terrible for polluting macro namespace
#else
    #include <sys/resource.h>
#endif

#include <stdio.h>
#include <iostream>
#include <algorithm>

#include <hipcub/hipcub.hpp>

#include "../test/test_util.h"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

/// Verbose output
bool g_verbose = false;

/// Timing iterations
int g_iterations = 100;

/// Default grid size
int g_grid_size = 1;

/// Uniform key samples
bool g_uniform_keys;


//---------------------------------------------------------------------
// Kernels
//---------------------------------------------------------------------

/**
 * Simple kernel for performing a block-wide sorting over integers
 */
template <
    typename    KeyType,
    int         BLOCK_THREADS,
    int         ITEMS_PER_THREAD>
__launch_bounds__ (BLOCK_THREADS)
__global__ void BlockSortKernel(
    KeyType     *d_in,          // Tile of input
    KeyType     *d_out,         // Tile of output
    clock_t     *d_elapsed)     // Elapsed cycle count of block scan
{
    enum { TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD };

    // Parameterize BlockRadixSort type for our thread block
    typedef BlockRadixSort<KeyType, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

    // Shared memory
    __shared__ typename BlockRadixSortT::SmemStorage smem_storage;

    // Per-thread tile items
    KeyType items[ITEMS_PER_THREAD];

    // Load items in striped fashion
    int block_offset = blockIdx.x * TILE_SIZE;
    BlockLoadDirectStriped(d_in + block_offset, items);

    // Start cycle timer
    clock_t start = clock();

    // Sort keys
    BlockRadixSortT::SortStriped(smem_storage, items);

    // Stop cycle timer
    clock_t stop = clock();

    // Store output
    BlockStoreDirectStriped(d_out + block_offset, items);

    // Store elapsed clocks
    if (threadIdx.x == 0)
    {
        d_elapsed[blockIdx.x] = (start > stop) ? start - stop : stop - start;
    }
}



//---------------------------------------------------------------------
// Host utilities
//---------------------------------------------------------------------


/**
 * Initialize sorting problem (and solution).
 */
template <typename KeyType>
void Initialize(
    KeyType *h_in,
    KeyType *h_reference,
    int num_items,
    int tile_size)
{
    for (int i = 0; i < num_items; ++i)
    {
        if (g_uniform_keys)
        {
            h_in[i] = 0;
        }
        else
        {
            RandomBits(h_in[i]);
        }
        h_reference[i] = h_in[i];
    }

    // Only sort the first tile
    std::sort(h_reference, h_reference + tile_size);
}


/**
 * Test BlockScan
 */
template <
    typename    KeyType,
    int         BLOCK_THREADS,
    int         ITEMS_PER_THREAD>
void Test()
{
    const int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

    // Allocate host arrays
    KeyType *h_in           = new KeyType[TILE_SIZE * g_grid_size];
    KeyType *h_reference    = new KeyType[TILE_SIZE * g_grid_size];
    clock_t *h_elapsed      = new clock_t[g_grid_size];

    // Initialize problem and reference output on host
    Initialize(h_in, h_reference, TILE_SIZE * g_grid_size, TILE_SIZE);

    // Initialize device arrays
    KeyType *d_in       = NULL;
    KeyType *d_out      = NULL;
    clock_t *d_elapsed  = NULL;
    CubDebugExit(hipMalloc((void**)&d_in,          sizeof(KeyType) * TILE_SIZE * g_grid_size));
    CubDebugExit(hipMalloc((void**)&d_out,         sizeof(KeyType) * TILE_SIZE * g_grid_size));
    CubDebugExit(hipMalloc((void**)&d_elapsed,     sizeof(clock_t) * g_grid_size));

    // Display input problem data
    if (g_verbose)
    {
        printf("Input data: ");
        for (int i = 0; i < TILE_SIZE; i++)
            std::cout << h_in[i] << ", ";
        printf("\n\n");
    }

    // CUDA device props
    Device device;
    int max_sm_occupancy;
    CubDebugExit(device.Init());
    CubDebugExit(device.MaxSmOccupancy(max_sm_occupancy, BlockSortKernel<KeyType, BLOCK_THREADS, ITEMS_PER_THREAD>, BLOCK_THREADS));

    // Copy problem to device
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(KeyType) * TILE_SIZE * g_grid_size, hipMemcpyHostToDevice));

    printf("BlockRadixSort %d items (%d timing iterations, %d blocks, %d threads, %d items per thread, %d SM occupancy):\n",
        TILE_SIZE * g_grid_size, g_iterations, g_grid_size, BLOCK_THREADS, ITEMS_PER_THREAD, max_sm_occupancy);
    fflush(stdout);

    // Run kernel once to prime caches and check result
    BlockSortKernel<KeyType, BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>>(
        d_in,
        d_out,
        d_elapsed);

    // Check for kernel errors and STDIO from the kernel, if any
    CubDebugExit(hipDeviceSynchronize());

    // Check results
    printf("\tOutput items: ");
    int compare = CompareDeviceResults(h_reference, d_out, TILE_SIZE, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);
    fflush(stdout);

    // Run this several times and average the performance results
    GpuTimer            timer;
    float               elapsed_millis          = 0.0;
    unsigned long long  elapsed_clocks          = 0;

    for (int i = 0; i < g_iterations; ++i)
    {
        timer.Start();

        // Run kernel
        BlockSortKernel<KeyType, BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>>(
            d_in,
            d_out,
            d_elapsed);

        timer.Stop();
        elapsed_millis += timer.ElapsedMillis();

        // Copy clocks from device
        CubDebugExit(hipMemcpy(h_elapsed, d_elapsed, sizeof(clock_t) * g_grid_size, hipMemcpyDeviceToHost));
        for (int i = 0; i < g_grid_size; i++)
            elapsed_clocks += h_elapsed[i];
    }

    // Check for kernel errors and STDIO from the kernel, if any
    CubDebugExit(hipDeviceSynchronize());

    // Display timing results
    float avg_millis            = elapsed_millis / g_iterations;
    float avg_items_per_sec     = float(TILE_SIZE * g_grid_size) / avg_millis / 1000.0;
    double avg_clocks           = double(elapsed_clocks) / g_iterations / g_grid_size;
    double avg_clocks_per_item  = avg_clocks / TILE_SIZE;

    printf("\tAverage BlockRadixSort::SortBlocked clocks: %.3f\n", avg_clocks);
    printf("\tAverage BlockRadixSort::SortBlocked clocks per item: %.3f\n", avg_clocks_per_item);
    printf("\tAverage kernel millis: %.4f\n", avg_millis);
    printf("\tAverage million items / sec: %.4f\n", avg_items_per_sec);
    fflush(stdout);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (h_elapsed) delete[] h_elapsed;
    if (d_in) CubDebugExit(hipFree(d_in));
    if (d_out) CubDebugExit(hipFree(d_out));
    if (d_elapsed) CubDebugExit(hipFree(d_elapsed));
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    g_uniform_keys = args.CheckCmdLineFlag("uniform");
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("grid-size", g_grid_size);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--i=<timing iterations (default:%d)>]"
            "[--grid-size=<grid size (default:%d)>]"
            "[--v] "
            "\n", argv[0], g_iterations, g_grid_size);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());
    fflush(stdout);

    // Run tests
    printf("\nuint32:\n"); fflush(stdout);
    Test<unsigned int, 128, 17>();
    printf("\n"); fflush(stdout);

    printf("\nfp32:\n"); fflush(stdout);
    Test<float, 128, 17>();
    printf("\n"); fflush(stdout);

    printf("\nuint8:\n"); fflush(stdout);
    Test<unsigned char, 128, 17>();
    printf("\n"); fflush(stdout);

    return 0;
}

