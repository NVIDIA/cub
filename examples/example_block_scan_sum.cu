#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple demonstration of hipcub::BlockScan
 *
 * Example compilation string:
 *
 * nvcc example_block_scan_sum.cu -gencode=arch=compute_20,code=\"sm_20,compute_20\" -o example_block_scan_sum
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console (define before including cub.h)
#define HIPCUB_STDERR

#include <stdio.h>
#include <iostream>

#include <hipcub/hipcub.hpp>

#include "../test/test_util.h"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

/// Verbose output
bool g_verbose = false;

/// Timing iterations
int g_iterations = 100;

/// Default grid size
int g_grid_size = 1;



//---------------------------------------------------------------------
// Kernels
//---------------------------------------------------------------------

/**
 * Simple kernel for performing a block-wide exclusive prefix sum over integers
 */
template <
    int         BLOCK_THREADS,
    int         ITEMS_PER_THREAD>
__global__ void BlockPrefixSumKernel(
    int         *d_in,          // Tile of input
    int         *d_out,         // Tile of output
    clock_t     *d_elapsed)     // Elapsed cycle count of block scan
{
    // Parameterize BlockScan type for our thread block
    typedef BlockScan<int, BLOCK_THREADS> BlockScanT;

    // Shared memory
    __shared__ typename BlockScanT::SmemStorage smem_storage;

    // Per-thread tile data
    int data[ITEMS_PER_THREAD];
    BlockLoadVectorized(d_in, data);

    // Start cycle timer
    clock_t start = clock();

    // Compute exclusive prefix sum
    int aggregate;
    BlockScanT::ExclusiveSum(smem_storage, data, data, aggregate);

    // Stop cycle timer
    clock_t stop = clock();

    // Store output
    BlockStoreVectorized(d_out, data);

    // Store aggregate and elapsed clocks
    if (threadIdx.x == 0)
    {
        *d_elapsed = (start > stop) ? start - stop : stop - start;
        d_out[BLOCK_THREADS * ITEMS_PER_THREAD] = aggregate;
    }
}



//---------------------------------------------------------------------
// Host utilities
//---------------------------------------------------------------------

/**
 * Initialize exclusive prefix sum problem (and solution).
 * Returns the aggregate
 */
int Initialize(
    int *h_in,
    int *h_reference,
    int num_items)
{
    int inclusive = 0;

    for (int i = 0; i < num_items; ++i)
    {
        h_in[i] = i % 17;

        h_reference[i] = inclusive;
        inclusive += h_in[i];
    }

    return inclusive;
}


/**
 * Test thread block scan
 */
template <
    int BLOCK_THREADS,
    int ITEMS_PER_THREAD>
void Test()
{
    const int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

    // Allocate host arrays
    int *h_in           = new int[TILE_SIZE];
    int *h_reference    = new int[TILE_SIZE];
    int *h_gpu          = new int[TILE_SIZE + 1];

    // Initialize problem and reference output on host
    int h_aggregate = Initialize(h_in, h_reference, TILE_SIZE);

    // Initialize device arrays
    int *d_in           = NULL;
    int *d_out          = NULL;
    clock_t *d_elapsed  = NULL;
    hipMalloc((void**)&d_in,          sizeof(int) * TILE_SIZE);
    hipMalloc((void**)&d_out,         sizeof(int) * (TILE_SIZE + 1));
    hipMalloc((void**)&d_elapsed,     sizeof(clock_t));

    // Display input problem data
    if (g_verbose)
    {
        printf("Input data: ");
        for (int i = 0; i < TILE_SIZE; i++)
            printf("%d, ", h_in[i]);
        printf("\n\n");
    }

    // CUDA device props
    Device device;
    int max_sm_occupancy;
    CubDebugExit(device.Init());
    CubDebugExit(device.MaxSmOccupancy(max_sm_occupancy, BlockPrefixSumKernel<BLOCK_THREADS, ITEMS_PER_THREAD>, BLOCK_THREADS));

    // Copy problem to device
    hipMemcpy(d_in, h_in, sizeof(int) * TILE_SIZE, hipMemcpyHostToDevice);

    printf("BlockScan %d items (%d timing iterations, %d blocks, %d threads, %d items per thread, %d SM occupancy):\n",
        TILE_SIZE, g_iterations, g_grid_size, BLOCK_THREADS, ITEMS_PER_THREAD, max_sm_occupancy);

    // Run aggregate/prefix kernel
    BlockPrefixSumKernel<BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>>(
        d_in,
        d_out,
        d_elapsed);

    // Check results
    printf("\tOutput items: ");
    int compare = CompareDeviceResults(h_reference, d_out, TILE_SIZE, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Check total aggregate
    printf("\tAggregate: ");
    compare = CompareDeviceResults(&h_aggregate, d_out + TILE_SIZE, 1, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Run this several times and average the performance results
    GpuTimer    timer;
    float       elapsed_millis          = 0.0;
    clock_t     elapsed_clocks          = 0;

    for (int i = 0; i < g_iterations; ++i)
    {
        // Copy problem to device
        hipMemcpy(d_in, h_in, sizeof(int) * TILE_SIZE, hipMemcpyHostToDevice);

        timer.Start();

        // Run aggregate/prefix kernel
        BlockPrefixSumKernel<BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>>(
            d_in,
            d_out,
            d_elapsed);

        timer.Stop();
        elapsed_millis += timer.ElapsedMillis();

        // Copy clocks from device
        clock_t clocks;
        CubDebugExit(hipMemcpy(&clocks, d_elapsed, sizeof(clock_t), hipMemcpyDeviceToHost));
        elapsed_clocks += clocks;

    }

    // Check for kernel errors and STDIO from the kernel, if any
    CubDebugExit(hipDeviceSynchronize());

    // Display timing results
    float avg_millis            = elapsed_millis / g_iterations;
    float avg_items_per_sec     = float(TILE_SIZE * g_grid_size) / avg_millis / 1000.0;
    float avg_clocks            = float(elapsed_clocks) / g_iterations;
    float avg_clocks_per_item   = avg_clocks / TILE_SIZE;

    printf("\tAverage BlockRadixSort::SortBlocked clocks: %.3f\n", avg_clocks);
    printf("\tAverage BlockRadixSort::SortBlocked clocks per item: %.3f\n", avg_clocks_per_item);
    printf("\tAverage kernel millis: %.4f\n", avg_millis);
    printf("\tAverage million items / sec: %.4f\n", avg_items_per_sec);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (h_gpu) delete[] h_gpu;
    if (d_in) hipFree(d_in);
    if (d_out) hipFree(d_out);
    if (d_elapsed) hipFree(d_elapsed);
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("grid-size", g_grid_size);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--i=<timing iterations (default:%d)>]"
            "[--grid-size=<grid size (default:%d)>]"
            "[--v] "
            "\n", argv[0], g_iterations, g_grid_size);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());


/** Add tests here **/

    // Run tests
    Test<1024, 1>();
    Test<512, 2>();
    Test<256, 4>();
    Test<128, 8>();
    Test<64, 16>();
    Test<32, 32>();
    Test<16, 64>();

/****/

    return 0;
}

