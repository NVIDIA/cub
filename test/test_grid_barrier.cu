#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test evaluation for software global barrier throughput
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>

#include <stdio.h>
#include <hipcub/hipcub.hpp>
#include <cub/grid/grid_barrier.cuh>
#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Kernel that iterates through the specified number of software global barriers
 */
__global__ void Kernel(
    GridBarrier global_barrier,
    int iterations)
{
    for (int i = 0; i < iterations; i++)
    {
        global_barrier.Sync();
    }
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;

    // Defaults
    int iterations = 10000;
    int block_size = 128;
    int grid_size = -1;

    // Initialize command line
    CommandLineArgs args(argc, argv);

    // Get args
    args.GetCmdLineArgument("i", iterations);
    args.GetCmdLineArgument("grid-size", grid_size);
    args.GetCmdLineArgument("block-size", block_size);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>]"
            "[--i=<iterations>]"
            "[--grid-size<grid-size>]"
            "[--block-size<block-size>]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Initialize CUDA device properties
    Device cuda_props;
    CubDebugExit(cuda_props.Init());

    // Compute grid size and occupancy
    int occupancy = CUB_MIN(
        (cuda_props.max_sm_threads / block_size),
        cuda_props.max_sm_blocks);

    if (grid_size == -1)
    {
        grid_size = occupancy * cuda_props.sm_count;
    }
    else
    {
        occupancy = grid_size / cuda_props.sm_count;
    }

    printf("Initializing software global barrier for Kernel<<<%d,%d>>> with %d occupancy\n",
        grid_size, block_size, occupancy);
    fflush(stdout);

    // Init global barrier
    GridBarrierLifetime global_barrier;
    global_barrier.Setup(grid_size);

    // Time kernel
    GpuTimer gpu_timer;
    gpu_timer.Start();
    Kernel<<<grid_size, block_size>>>(global_barrier, iterations);
    gpu_timer.Stop();

    retval = HipcubDebug(hipDeviceSynchronize());

    // Output timing results
    float avg_elapsed = gpu_timer.ElapsedMillis() / float(iterations);
    printf("%d iterations, %f total elapsed millis, %f avg elapsed millis\n",
        iterations,
        gpu_timer.ElapsedMillis(),
        avg_elapsed);

    return retval;
}
