#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of iterator utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iterator>
#include <stdio.h>
#include <typeinfo>

#include <cub/iterator/arg_index_input_iterator.cuh>
#include <cub/iterator/cache_modified_input_iterator.cuh>
#include <cub/iterator/cache_modified_output_iterator.cuh>
#include <cub/iterator/constant_input_iterator.cuh>
#include <cub/iterator/counting_input_iterator.cuh>
#include <cub/iterator/tex_obj_input_iterator.cuh>
#include <cub/iterator/transform_input_iterator.cuh>

#include <cub/util_type.cuh>
#include <cub/util_allocator.cuh>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;
CachingDeviceAllocator  g_allocator(true);

// Dispatch types
enum Backend
{
    CUB,        // CUB method
    CDP,        // GPU-based (dynamic parallelism) dispatch to CUB method
};


template <typename T>
struct TransformOp
{
    // Increment transform
    __host__ __device__ __forceinline__ T operator()(T input) const
    {
        T addend;
        InitValue(INTEGER_SEED, addend, 1);
        return input + addend;
    }
};

struct SelectOp
{
    template <typename T>
    __host__ __device__ __forceinline__ bool operator()(T input)
    {
        return true;
    }
};


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Test random access input iterator
 */
template <
    typename InputIteratorT,
    typename T>
__global__ void Kernel(
    InputIteratorT    d_in,
    T                 *d_out,
    InputIteratorT    *d_itrs)
{
    d_out[0] = *d_in;               // Value at offset 0
    d_out[1] = d_in[100];           // Value at offset 100
    d_out[2] = *(d_in + 1000);      // Value at offset 1000
    d_out[3] = *(d_in + 10000);     // Value at offset 10000

    d_in++;
    d_out[4] = d_in[0];             // Value at offset 1

    d_in += 20;
    d_out[5] = d_in[0];             // Value at offset 21
    d_itrs[0] = d_in;               // Iterator at offset 21

    d_in -= 10;
    d_out[6] = d_in[0];             // Value at offset 11;

    d_in -= 11;
    d_out[7] = d_in[0];             // Value at offset 0
    d_itrs[1] = d_in;               // Iterator at offset 0
}



//---------------------------------------------------------------------
// Host testing subroutines
//---------------------------------------------------------------------


/**
 * Run iterator test on device
 */
template <
    typename        InputIteratorT,
    typename        T,
    int             TEST_VALUES>
void Test(
    InputIteratorT  d_in,
    T               (&h_reference)[TEST_VALUES])
{
    // Allocate device arrays
    T                 *d_out    = NULL;
    InputIteratorT    *d_itrs   = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out,     sizeof(T) * TEST_VALUES));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_itrs,    sizeof(InputIteratorT) * 2));

    int compare;

    // Run unguarded kernel
    Kernel<<<1, 1>>>(d_in, d_out, d_itrs);

    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Check results
    compare = CompareDeviceResults(h_reference, d_out, TEST_VALUES, g_verbose, g_verbose);
    printf("\tValues: %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Check iterator at offset 21
    InputIteratorT h_itr = d_in + 21;
    compare = CompareDeviceResults(&h_itr, d_itrs, 1, g_verbose, g_verbose);
    printf("\tIterators: %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Check iterator at offset 0
    compare = CompareDeviceResults(&d_in, d_itrs + 1, 1, g_verbose, g_verbose);
    printf("\tIterators: %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_itrs) CubDebugExit(g_allocator.DeviceFree(d_itrs));
}


/**
 * Test constant iterator
 */
template <typename T>
void TestConstant(T base)
{
    printf("\nTesting constant iterator on type %s (base: %lld)\n", typeid(T).name(), (unsigned long long) (base)); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    T h_reference[8] = {base, base, base, base, base, base, base, base};
    ConstantInputIterator<T> d_itr(base);
    Test(d_itr, h_reference);
}


/**
 * Test counting iterator
 */
template <typename T>
void TestCounting(T base)
{
    printf("\nTesting counting iterator on type %s (base: %d) \n", typeid(T).name(), int(base)); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = static_cast<T>(base + 0);          // Value at offset 0
    h_reference[1] = static_cast<T>(base + 100);        // Value at offset 100
    h_reference[2] = static_cast<T>(base + 1000);       // Value at offset 1000
    h_reference[3] = static_cast<T>(base + 10000);      // Value at offset 10000
    h_reference[4] = static_cast<T>(base + 1);          // Value at offset 1
    h_reference[5] = static_cast<T>(base + 21);         // Value at offset 21
    h_reference[6] = static_cast<T>(base + 11);         // Value at offset 11
    h_reference[7] = static_cast<T>(base + 0);          // Value at offset 0;

    CountingInputIterator<T> d_itr(base);
    Test(d_itr, h_reference);
}


/**
 * Test modified iterator
 */
template <typename T, typename CastT>
void TestModified()
{
    printf("\nTesting cache-modified iterator on type %s\n", typeid(T).name()); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    constexpr int TEST_VALUES = 11000;

    T *h_data = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
    {
        RandomBits(h_data[i]);
    }

    // Allocate device arrays
    T *d_data = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(T) * TEST_VALUES));
    CubDebugExit(hipMemcpy(d_data, h_data, sizeof(T) * TEST_VALUES, hipMemcpyHostToDevice));

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = h_data[0];          // Value at offset 0
    h_reference[1] = h_data[100];        // Value at offset 100
    h_reference[2] = h_data[1000];       // Value at offset 1000
    h_reference[3] = h_data[10000];      // Value at offset 10000
    h_reference[4] = h_data[1];          // Value at offset 1
    h_reference[5] = h_data[21];         // Value at offset 21
    h_reference[6] = h_data[11];         // Value at offset 11
    h_reference[7] = h_data[0];          // Value at offset 0;

    Test(CacheModifiedInputIterator<LOAD_DEFAULT, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_CA, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_CG, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_CS, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_CV, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_LDG, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_VOLATILE, T>((CastT*) d_data), h_reference);

    if (h_data) delete[] h_data;
    if (d_data) CubDebugExit(g_allocator.DeviceFree(d_data));
}


/**
 * Test transform iterator
 */
template <typename T, typename CastT>
void TestTransform()
{
    printf("\nTesting transform iterator on type %s\n", typeid(T).name()); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    constexpr int TEST_VALUES = 11000;

    T *h_data = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
    {
        InitValue(INTEGER_SEED, h_data[i], i);
    }

    // Allocate device arrays
    T *d_data = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(T) * TEST_VALUES));
    CubDebugExit(hipMemcpy(d_data, h_data, sizeof(T) * TEST_VALUES, hipMemcpyHostToDevice));

    TransformOp<T> op;

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = op(h_data[0]);          // Value at offset 0
    h_reference[1] = op(h_data[100]);        // Value at offset 100
    h_reference[2] = op(h_data[1000]);       // Value at offset 1000
    h_reference[3] = op(h_data[10000]);      // Value at offset 10000
    h_reference[4] = op(h_data[1]);          // Value at offset 1
    h_reference[5] = op(h_data[21]);         // Value at offset 21
    h_reference[6] = op(h_data[11]);         // Value at offset 11
    h_reference[7] = op(h_data[0]);          // Value at offset 0;

    TransformInputIterator<T, TransformOp<T>, CastT*> d_itr((CastT*) d_data, op);
    Test(d_itr, h_reference);

    if (h_data) delete[] h_data;
    if (d_data) CubDebugExit(g_allocator.DeviceFree(d_data));
}


/**
 * Test tex-obj texture iterator
 */
template <typename T, typename CastT>
void TestTexObj()
{
    printf("\nTesting tex-obj iterator on type %s\n", typeid(T).name()); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    const unsigned int TEST_VALUES          = 11000;
    const unsigned int DUMMY_OFFSET         = 500;
    const unsigned int DUMMY_TEST_VALUES    = TEST_VALUES - DUMMY_OFFSET;

    T *h_data = new T[TEST_VALUES];
    for (unsigned int i = 0; i < TEST_VALUES; ++i)
    {
        RandomBits(h_data[i]);
    }

    // Allocate device arrays
    T *d_data   = NULL;
    T *d_dummy  = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(T) * TEST_VALUES));
    CubDebugExit(hipMemcpy(d_data, h_data, sizeof(T) * TEST_VALUES, hipMemcpyHostToDevice));

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_dummy, sizeof(T) * DUMMY_TEST_VALUES));
    CubDebugExit(hipMemcpy(d_dummy, h_data + DUMMY_OFFSET, sizeof(T) * DUMMY_TEST_VALUES, hipMemcpyHostToDevice));

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = h_data[0];          // Value at offset 0
    h_reference[1] = h_data[100];        // Value at offset 100
    h_reference[2] = h_data[1000];       // Value at offset 1000
    h_reference[3] = h_data[10000];      // Value at offset 10000
    h_reference[4] = h_data[1];          // Value at offset 1
    h_reference[5] = h_data[21];         // Value at offset 21
    h_reference[6] = h_data[11];         // Value at offset 11
    h_reference[7] = h_data[0];          // Value at offset 0;

    // Create and bind obj-based test iterator
    TexObjInputIterator<T> d_obj_itr;
    CubDebugExit(d_obj_itr.BindTexture((CastT*) d_data, sizeof(T) * TEST_VALUES));

    Test(d_obj_itr, h_reference);

    if (h_data) delete[] h_data;
    if (d_data) CubDebugExit(g_allocator.DeviceFree(d_data));
    if (d_dummy) CubDebugExit(g_allocator.DeviceFree(d_dummy));
}

/**
 * Test texture transform iterator
 */
template <typename T, typename CastT>
void TestTexTransform()
{
    printf("\nTesting tex-transform iterator on type %s\n", typeid(T).name()); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    constexpr int TEST_VALUES = 11000;

    T *h_data = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
    {
        InitValue(INTEGER_SEED, h_data[i], i);
    }

    // Allocate device arrays
    T *d_data = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(T) * TEST_VALUES));
    CubDebugExit(hipMemcpy(d_data, h_data, sizeof(T) * TEST_VALUES, hipMemcpyHostToDevice));

    TransformOp<T> op;

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = op(h_data[0]);          // Value at offset 0
    h_reference[1] = op(h_data[100]);        // Value at offset 100
    h_reference[2] = op(h_data[1000]);       // Value at offset 1000
    h_reference[3] = op(h_data[10000]);      // Value at offset 10000
    h_reference[4] = op(h_data[1]);          // Value at offset 1
    h_reference[5] = op(h_data[21]);         // Value at offset 21
    h_reference[6] = op(h_data[11]);         // Value at offset 11
    h_reference[7] = op(h_data[0]);          // Value at offset 0;

    // Create and bind texture iterator
    typedef TexObjInputIterator<T> TextureIterator;

    TextureIterator d_tex_itr;
    CubDebugExit(d_tex_itr.BindTexture((CastT*) d_data, sizeof(T) * TEST_VALUES));

    // Create transform iterator
    TransformInputIterator<T, TransformOp<T>, TextureIterator> xform_itr(d_tex_itr, op);

    Test(xform_itr, h_reference);

    CubDebugExit(d_tex_itr.UnbindTexture());
    if (h_data) delete[] h_data;
    if (d_data) CubDebugExit(g_allocator.DeviceFree(d_data));
}

/**
 * Run non-integer tests
 */
template <typename T, typename CastT>
void Test(Int2Type<false> /* is_integer */)
{
    TestModified<T, CastT>();
    TestTransform<T, CastT>();
    TestTexObj<T, CastT>();
    TestTexTransform<T, CastT>();
}

/**
 * Run integer tests
 */
template <typename T, typename CastT>
void Test(Int2Type<true> /* is_integer */)
{
    TestConstant<T>(0);
    TestConstant<T>(99);

    TestCounting<T>(0);
    TestCounting<T>(99);

    // Run non-integer tests
    Test<T, CastT>(Int2Type<false>());
}

/**
 * Run tests
 */
template <typename T>
void Test()
{
    enum {
        IS_INTEGER = (Traits<T>::CATEGORY == SIGNED_INTEGER) || (Traits<T>::CATEGORY == UNSIGNED_INTEGER)
    };

    // Test non-const type
    Test<T, T>(Int2Type<IS_INTEGER>());

    // Test non-const type
    Test<T, const T>(Int2Type<IS_INTEGER>());
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Get ptx version
    int ptx_version = 0;
    CubDebugExit(PtxVersion(ptx_version));

    // Evaluate different data types
    Test<char>();
    Test<short>();
    Test<int>();
    Test<long>();
    Test<long long>();
    Test<float>();
    Test<double>();

    Test<char2>();
    Test<short2>();
    Test<int2>();
    Test<long2>();
    Test<longlong2>();
    Test<float2>();
    Test<double2>();

    Test<char3>();
    Test<short3>();
    Test<int3>();
    Test<long3>();
    Test<longlong3>();
    Test<float3>();
    Test<double3>();

    Test<char4>();
    Test<short4>();
    Test<int4>();
    Test<long4>();
    Test<longlong4>();
    Test<float4>();
    Test<double4>();

    Test<TestFoo>();
    Test<TestBar>();

    printf("\nTest complete\n"); fflush(stdout);

    return 0;
}



