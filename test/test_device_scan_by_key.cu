#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2021 NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of DeviceScan utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <typeinfo>

#include <cub/util_allocator.cuh>
#include <cub/iterator/constant_input_iterator.cuh>
#include <cub/iterator/discard_output_iterator.cuh>
#include <cub/iterator/counting_input_iterator.cuh>
#include <cub/iterator/transform_input_iterator.cuh>
#include <cub/device/device_scan.cuh>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose           = false;
int                     g_timing_iterations = 0;
double                  g_device_giga_bandwidth;
CachingDeviceAllocator  g_allocator(true);

// Dispatch types
enum Backend
{
    CUB,        // CUB method
    CDP,        // GPU-based (dynamic parallelism) dispatch to CUB method
};


enum AliasMode
{
  AliasNone,  // output is allocated
  AliasKeys,  // output is an alias of input keys
  AliasValues // output is an alias of input values
};


/**
 * \brief WrapperFunctor (for precluding test-specialized dispatch to *Sum variants)
 */
template<typename OpT>
struct WrapperFunctor
{
    OpT op;

    WrapperFunctor(OpT op) : op(op) {}

    template <typename T>
    __host__ __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return op(a, b);
    }
};

/**
 * \brief DivideByFiveFunctor (used by TestIterator)
 */
template<typename OutputT>
struct DivideByFiveFunctor
{
    template <typename T>
    __host__ __device__ __forceinline__ OutputT operator()(const T &a) const
    {
        return static_cast<OutputT>(a / 5);
    }
};

/**
 * \brief Mod2Equality (used for non-bool keys to make keys more likely to equal each other)
 */
struct Mod2Equality
{
    template <typename T>
    __host__ __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return (a % 2) == (b % 2);
    }
};


//---------------------------------------------------------------------
// Dispatch to different CUB DeviceScan entrypoints
//---------------------------------------------------------------------

/**
 * Dispatch to exclusive scan entrypoint
 */
template <typename IsPrimitiveT, typename KeysInputIteratorT, typename ValuesInputIteratorT, typename ValuesOutputIteratorT, typename ScanOpT, typename InitialValueT, typename OffsetT, typename EqualityOpT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CUB>         /*dispatch_to*/,
    IsPrimitiveT          /*is_primitive*/,
    int                   timing_timing_iterations,
    size_t                */*d_temp_storage_bytes*/,
    hipError_t           */*d_cdp_error*/,

    void*                 d_temp_storage,
    size_t&               temp_storage_bytes,
    KeysInputIteratorT    d_keys_in,
    ValuesInputIteratorT  d_values_in,
    ValuesOutputIteratorT d_values_out,
    ScanOpT               scan_op,
    InitialValueT         initial_value,
    OffsetT               num_items,
    EqualityOpT           equality_op,
    hipStream_t          stream,
    bool                  debug_synchronous)
{
    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceScan::ExclusiveScanByKey(d_temp_storage, temp_storage_bytes, d_keys_in, d_values_in, d_values_out, scan_op, initial_value, num_items, equality_op, stream, debug_synchronous);
    }
    return error;
}


/**
 * Dispatch to exclusive sum entrypoint
 */
template <typename KeysInputIteratorT, typename ValuesInputIteratorT, typename ValuesOutputIteratorT, typename InitialValueT, typename OffsetT, typename EqualityOpT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CUB>         /*dispatch_to*/,
    Int2Type<true>        /*is_primitive*/,
    int                   timing_timing_iterations,
    size_t                */*d_temp_storage_bytes*/,
    hipError_t           */*d_cdp_error*/,

    void*                 d_temp_storage,
    size_t&               temp_storage_bytes,
    KeysInputIteratorT    d_keys_in,
    ValuesInputIteratorT  d_values_in,
    ValuesOutputIteratorT d_values_out,
    Sum                   /*scan_op*/,
    InitialValueT         /*initial_value*/,
    OffsetT               num_items,
    EqualityOpT           equality_op,
    hipStream_t          stream,
    bool                  debug_synchronous)
{
    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceScan::ExclusiveSumByKey(d_temp_storage, temp_storage_bytes, d_keys_in, d_values_in, d_values_out, num_items, equality_op, stream, debug_synchronous);
    }
    return error;
}


/**
 * Dispatch to inclusive scan entrypoint
 */
template <typename IsPrimitiveT, typename KeysInputIteratorT, typename ValuesInputIteratorT, typename ValuesOutputIteratorT, typename ScanOpT, typename OffsetT, typename EqualityOpT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CUB>         /*dispatch_to*/,
    IsPrimitiveT          /*is_primitive*/,
    int                   timing_timing_iterations,
    size_t                */*d_temp_storage_bytes*/,
    hipError_t           */*d_cdp_error*/,

    void*                 d_temp_storage,
    size_t&               temp_storage_bytes,
    KeysInputIteratorT    d_keys_in,
    ValuesInputIteratorT  d_values_in,
    ValuesOutputIteratorT d_values_out,
    ScanOpT               scan_op,
    NullType              /*initial_value*/,
    OffsetT               num_items,
    EqualityOpT           equality_op,
    hipStream_t          stream,
    bool                  debug_synchronous)
{
    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceScan::InclusiveScanByKey(d_temp_storage, temp_storage_bytes, d_keys_in, d_values_in, d_values_out, scan_op, num_items, equality_op, stream, debug_synchronous);
    }
    return error;
}


/**
 * Dispatch to inclusive sum entrypoint
 */
template <typename KeysInputIteratorT, typename ValuesInputIteratorT, typename ValuesOutputIteratorT, typename OffsetT, typename EqualityOpT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CUB>         /*dispatch_to*/,
    Int2Type<true>        /*is_primitive*/,
    int                   timing_timing_iterations,
    size_t                */*d_temp_storage_bytes*/,
    hipError_t           */*d_cdp_error*/,

    void*                 d_temp_storage,
    size_t&               temp_storage_bytes,
    KeysInputIteratorT    d_keys_in,
    ValuesInputIteratorT  d_values_in,
    ValuesOutputIteratorT d_values_out,
    Sum                   /*scan_op*/,
    NullType              /*initial_value*/,
    OffsetT               num_items,
    EqualityOpT           equality_op,
    hipStream_t          stream,
    bool                  debug_synchronous)
{
    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceScan::InclusiveSumByKey(d_temp_storage, temp_storage_bytes, d_keys_in, d_values_in, d_values_out, num_items, equality_op, stream, debug_synchronous);
    }
    return error;
}

//---------------------------------------------------------------------
// CUDA Nested Parallelism Test Kernel
//---------------------------------------------------------------------

/**
 * Simple wrapper kernel to invoke DeviceScan
 */
template <typename IsPrimitiveT, typename KeysInputIteratorT, typename ValuesInputIteratorT, typename ValuesOutputIteratorT, typename ScanOpT, typename InitialValueT, typename OffsetT, typename EqualityOpT>
__global__ void CnpDispatchKernel(
    IsPrimitiveT          is_primitive,
    int                   timing_timing_iterations,
    size_t                *d_temp_storage_bytes,
    hipError_t           *d_cdp_error,

    void*                 d_temp_storage,
    size_t                temp_storage_bytes,
    KeysInputIteratorT    d_keys_in,
    ValuesInputIteratorT  d_values_in,
    ValuesOutputIteratorT d_values_out,
    ScanOpT               scan_op,
    InitialValueT         initial_value,
    OffsetT               num_items,
    EqualityOpT           equality_op,
    bool                  debug_synchronous)
{
#ifndef CUB_CDP
    (void)is_primitive;
    (void)timing_timing_iterations;
    (void)d_temp_storage_bytes;
    (void)d_cdp_error;
    (void)d_temp_storage;
    (void)temp_storage_bytes;
    (void)d_keys_in;
    (void)d_values_in;
    (void)d_values_out;
    (void)scan_op;
    (void)initial_value;
    (void)num_items;
    (void)equality_op;
    (void)debug_synchronous;
    *d_cdp_error = hipErrorNotSupported;
#else
    *d_cdp_error = Dispatch(
        Int2Type<CUB>(),
        is_primitive,
        timing_timing_iterations,
        d_temp_storage_bytes,
        d_cdp_error,
        d_temp_storage,
        temp_storage_bytes,
        d_in,
        d_out,
        scan_op,
        initial_value,
        num_items,
        0,
        debug_synchronous);

    *d_temp_storage_bytes = temp_storage_bytes;
#endif
}


/**
 * Dispatch to CDP kernel
 */
template <typename IsPrimitiveT, typename KeysInputIteratorT, typename ValuesInputIteratorT, typename ValuesOutputIteratorT, typename ScanOpT, typename InitialValueT, typename OffsetT, typename EqualityOpT>
hipError_t Dispatch(
    Int2Type<CDP>         dispatch_to,
    IsPrimitiveT          is_primitive,
    int                   timing_timing_iterations,
    size_t                *d_temp_storage_bytes,
    hipError_t           *d_cdp_error,

    void*                 d_temp_storage,
    size_t&               temp_storage_bytes,
    KeysInputIteratorT    d_keys_in,
    ValuesInputIteratorT  d_values_in,
    ValuesOutputIteratorT d_values_out,
    ScanOpT               scan_op,
    InitialValueT         initial_value,
    OffsetT               num_items,
    EqualityOpT           equality_op,
    hipStream_t          stream,
    bool                  debug_synchronous)
{
    // Invoke kernel to invoke device-side dispatch
    CnpDispatchKernel<<<1,1>>>(
        is_primitive,
        timing_timing_iterations,
        d_temp_storage_bytes,
        d_cdp_error,
        d_temp_storage,
        temp_storage_bytes,
        d_keys_in,
        d_values_in,
        d_values_out,
        scan_op,
        initial_value,
        equality_op,
        num_items,
        debug_synchronous);

    // Copy out temp_storage_bytes
    CubDebugExit(hipMemcpy(&temp_storage_bytes, d_temp_storage_bytes, sizeof(size_t) * 1, hipMemcpyDeviceToHost));

    // Copy out error
    hipError_t retval;
    CubDebugExit(hipMemcpy(&retval, d_cdp_error, sizeof(hipError_t) * 1, hipMemcpyDeviceToHost));
    return retval;
}


//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------


/**
 * Initialize problem
 */
template <typename T>
void Initialize(
    GenMode      gen_mode,
    T            *h_in,
    int          num_items)
{
    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
    }

    if (g_verbose)
    {
        printf("Input:\n");
        DisplayResults(h_in, num_items);
        printf("\n\n");
    }
}

/**
 * Solve exclusive-scan problem
 */
template <
    typename        KeysInputIteratorT,
    typename        ValuesInputIteratorT,
    typename        OutputT,
    typename        ScanOpT,
    typename        InitialValueT,
    typename        EqualityOpT>
void Solve(
    KeysInputIteratorT    h_keys_in,
    ValuesInputIteratorT  h_values_in,
    OutputT               *h_reference,
    int                   num_items,
    ScanOpT               scan_op,
    InitialValueT         initial_value,
    EqualityOpT           equality_op)
{
    // Use the initial value type for accumulation per P0571
    using AccumT = InitialValueT;

    if (num_items > 0)
    {
        for (int i = 0; i < num_items;) {
            AccumT val         = static_cast<AccumT>(h_values_in[i]);
            h_reference[i]     = initial_value;
            AccumT inclusive   = scan_op(initial_value, val);

            ++i;

            for (; i < num_items && equality_op(h_keys_in[i - 1], h_keys_in[i]); ++i)
            {
                val = static_cast<AccumT>(h_values_in[i]);
                h_reference[i] = static_cast<OutputT>(inclusive);
                inclusive = scan_op(inclusive, val);
            }
        }
    }
}


/**
 * Solve inclusive-scan problem
 */
template <
    typename        KeysInputIteratorT,
    typename        ValuesInputIteratorT,
    typename        OutputT,
    typename        ScanOpT,
    typename        EqualityOpT>
void Solve(
    KeysInputIteratorT    h_keys_in,
    ValuesInputIteratorT  h_values_in,
    OutputT               *h_reference,
    int                   num_items,
    ScanOpT               scan_op,
    NullType              /*initial_value*/,
    EqualityOpT           equality_op)
{
    // When no initial value type is supplied, use InputT for accumulation
    // per P0571
    using AccumT = hipcub::detail::value_t<ValuesInputIteratorT>;

    if (num_items > 0)
    {
        for (int i = 0; i < num_items;) {
            AccumT inclusive    = h_values_in[i];
            h_reference[i]      = static_cast<OutputT>(inclusive);

            ++i;

            for (; i < num_items && equality_op(h_keys_in[i - 1], h_keys_in[i]); ++i)
            {
                AccumT val = h_values_in[i];
                inclusive = scan_op(inclusive, val);
                h_reference[i] = static_cast<OutputT>(inclusive);
            }
        }
    }
}

template<typename OutputT, typename DeviceInputIteratorT, bool InPlace>
struct AllocateOutput {
    static void run(OutputT *&d_out, DeviceInputIteratorT, int num_items) {
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(OutputT) * num_items));
    }
};

template<typename OutputT>
struct AllocateOutput<OutputT, OutputT *, true> {
    static void run(OutputT *&d_out, OutputT *d_in, int /* num_items */) {
        d_out = d_in;
    }
};

/**
 * Test DeviceScan for a given problem input
 */
template <
    Backend             BACKEND,
    typename            KeysInputIteratorT,
    typename            ValuesInputIteratorT,
    typename            OutputT,
    typename            ScanOpT,
    typename            InitialValueT,
    typename            EqualityOpT,
    AliasMode           Mode=AliasNone>
void Test(
    KeysInputIteratorT      d_keys_in,
    ValuesInputIteratorT    d_values_in,
    OutputT                 *h_reference,
    int                     num_items,
    ScanOpT                 scan_op,
    InitialValueT           initial_value,
    EqualityOpT             equality_op)
{
    using KeyT = hipcub::detail::value_t<KeysInputIteratorT>;
    using InputT = hipcub::detail::value_t<ValuesInputIteratorT>;

    // Allocate device output array
    OutputT *d_values_out = NULL;

    if (Mode == AliasKeys)
    {
      AllocateOutput<OutputT, KeysInputIteratorT, Mode == AliasKeys>::run(
        d_values_out,
        d_keys_in,
        num_items);
    }
    else
    {
      AllocateOutput<OutputT, ValuesInputIteratorT, Mode == AliasValues>::run(
        d_values_out,
        d_values_in,
        num_items);
    }

    // Allocate CDP device arrays
    size_t          *d_temp_storage_bytes = NULL;
    hipError_t     *d_cdp_error = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_temp_storage_bytes,  sizeof(size_t) * 1));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_cdp_error,   sizeof(hipError_t) * 1));

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;
    CubDebugExit(Dispatch(
        Int2Type<BACKEND>(),
        Int2Type<Traits<OutputT>::PRIMITIVE>(),
        1,
        d_temp_storage_bytes,
        d_cdp_error,
        d_temp_storage,
        temp_storage_bytes,
        d_keys_in,
        d_values_in,
        d_values_out,
        scan_op,
        initial_value,
        num_items,
        equality_op,
        0,
        true));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Clear device output array
    if (Mode == AliasNone)
    {
      CubDebugExit(hipMemset(d_values_out, 0, sizeof(OutputT) * num_items));
    }

    // Run warmup/correctness iteration
    CubDebugExit(Dispatch(
        Int2Type<BACKEND>(),
        Int2Type<Traits<OutputT>::PRIMITIVE>(),
        1,
        d_temp_storage_bytes,
        d_cdp_error,
        d_temp_storage,
        temp_storage_bytes,
        d_keys_in,
        d_values_in,
        d_values_out,
        scan_op,
        initial_value,
        num_items,
        equality_op,
        0,
        true));

    // Check for correctness (and display results, if specified)
    const int compare = CompareDeviceResults(h_reference,
                                             d_values_out,
                                             num_items,
                                             true,
                                             g_verbose);

    printf("\t%s", compare ? "FAIL" : "PASS");

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Display performance
    if (g_timing_iterations > 0)
    {
      // Performance
      GpuTimer gpu_timer;
      gpu_timer.Start();
      CubDebugExit(Dispatch(Int2Type<BACKEND>(),
          Int2Type<Traits<OutputT>::PRIMITIVE>(),
          g_timing_iterations,
          d_temp_storage_bytes,
          d_cdp_error,
          d_temp_storage,
          temp_storage_bytes,
          d_keys_in,
          d_values_in,
          d_values_out,
          scan_op,
          initial_value,
          num_items,
          equality_op,
          0,
          false));

      gpu_timer.Stop();
      float elapsed_millis = gpu_timer.ElapsedMillis();
        float avg_millis = elapsed_millis / g_timing_iterations;
        float giga_rate = float(num_items) / avg_millis / 1000.0f / 1000.0f;
        float giga_bandwidth = giga_rate * (sizeof(InputT) + sizeof(OutputT));
        printf(", %.3f avg ms, %.3f billion items/s, %.3f logical GB/s, %.1f%% peak",
            avg_millis, giga_rate, giga_bandwidth, giga_bandwidth / g_device_giga_bandwidth * 100.0);
    }

    printf("\n\n");

    // Cleanup
    if (Mode == AliasNone)
    {
      if (d_values_out) 
      {
        CubDebugExit(g_allocator.DeviceFree(d_values_out));
      }
    }

    if (d_temp_storage_bytes) CubDebugExit(g_allocator.DeviceFree(d_temp_storage_bytes));
    if (d_cdp_error) CubDebugExit(g_allocator.DeviceFree(d_cdp_error));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    // Correctness asserts
    AssertEquals(0, compare);
}

template <Backend BACKEND,
          typename KeysInputIteratorT,
          typename OutputT,
          typename ScanOpT,
          typename InitialValueT,
          typename EqualityOpT>
void TestInplaceValues(KeysInputIteratorT d_keys_in,
                       OutputT *d_values_in,
                       OutputT *h_reference,
                       int num_items,
                       ScanOpT scan_op,
                       InitialValueT initial_value,
                       EqualityOpT equality_op)
{
  Test<BACKEND,
       KeysInputIteratorT,
       OutputT *,
       OutputT,
       ScanOpT,
       InitialValueT,
       EqualityOpT,
       AliasValues>(d_keys_in,
                    d_values_in,
                    h_reference,
                    num_items,
                    scan_op,
                    initial_value,
                    equality_op);
}

template <Backend BACKEND,
          typename KeysInputIteratorT,
          typename ValuesInputIteratorT,
          typename OutputT,
          typename ScanOpT,
          typename InitialValueT,
          typename EqualityOpT>
void TestInplaceValues(KeysInputIteratorT,
                       ValuesInputIteratorT,
                       OutputT *,
                       int,
                       ScanOpT,
                       InitialValueT,
                       EqualityOpT)
{}

template <Backend BACKEND,
          typename T,
          typename ValuesInputIteratorT,
          typename ScanOpT,
          typename InitialValueT,
          typename EqualityOpT>
void TestInplaceKeys(T *d_keys_in,
                     ValuesInputIteratorT d_values_in,
                     T *h_reference,
                     int num_items,
                     ScanOpT scan_op,
                     InitialValueT initial_value,
                     EqualityOpT equality_op)
{
  Test<BACKEND,
       T *,
       ValuesInputIteratorT,
       T,
       ScanOpT,
       InitialValueT,
       EqualityOpT,
       AliasKeys>(d_keys_in,
                  d_values_in,
                  h_reference,
                  num_items,
                  scan_op,
                  initial_value,
                  equality_op);
}

template <Backend BACKEND,
          typename KeysInputIteratorT,
          typename ValuesInputIteratorT,
          typename OutputT,
          typename ScanOpT,
          typename InitialValueT,
          typename EqualityOpT>
void TestInplaceKeys(KeysInputIteratorT,
                     ValuesInputIteratorT,
                     OutputT *,
                     int,
                     ScanOpT,
                     InitialValueT,
                     EqualityOpT)
{}

/**
 * Test DeviceScan on pointer type
 */
template <
    Backend         BACKEND,
    typename        KeyT,
    typename        InputT,
    typename        OutputT,
    typename        ScanOpT,
    typename        InitialValueT,
    typename        EqualityOpT>
void TestPointer(
    int             num_items,
    GenMode         gen_mode,
    ScanOpT         scan_op,
    InitialValueT   initial_value,
    EqualityOpT     equality_op)
{
    printf("\nPointer %s %s hipcub::DeviceScan::%s %d items, %s->%s (%d->%d bytes) , gen-mode %s\n",
        (BACKEND == CDP) ? "CDP CUB" : "CUB",
        (std::is_same<InitialValueT, NullType>::value) ? "Inclusive" : "Exclusive",
        (std::is_same<ScanOpT, Sum>::value) ? "Sum" : "Scan",
        num_items,
        typeid(InputT).name(), typeid(OutputT).name(), (int) sizeof(InputT), (int) sizeof(OutputT),
        (gen_mode == RANDOM) ? "RANDOM" : (gen_mode == INTEGER_SEED) ? "SEQUENTIAL" : "HOMOGENOUS");
    fflush(stdout);

    // Allocate host arrays
    KeyT*       h_keys_in   = new KeyT[num_items];
    InputT*     h_values_in = new InputT[num_items];
    OutputT*    h_reference = new OutputT[num_items];

    // Initialize problem and solution
    Initialize(gen_mode, h_keys_in, num_items);
    Initialize(gen_mode, h_values_in, num_items);

    // If the output type is primitive and the operator is hipcub::Sum, the test
    // dispatcher throws away scan_op and initial_value for exclusive scan.
    // Without an initial_value arg, the accumulator switches to the input value
    // type.
    // Do the same thing here:
    if (Traits<OutputT>::PRIMITIVE &&
        std::is_same<ScanOpT, hipcub::Sum>::value &&
        !std::is_same<InitialValueT, NullType>::value)
    {
      Solve(h_keys_in, h_values_in, h_reference, num_items, hipcub::Sum{}, InputT{}, equality_op);
    }
    else
    {
      Solve(h_keys_in, h_values_in, h_reference, num_items, scan_op, initial_value, equality_op);
    }

    // Allocate problem device arrays
    KeyT *d_keys_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys_in, sizeof(KeyT) * num_items));
    InputT *d_values_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values_in, sizeof(InputT) * num_items));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_keys_in, h_keys_in, sizeof(KeyT) * num_items, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_values_in, h_values_in, sizeof(InputT) * num_items, hipMemcpyHostToDevice));

    // Run Test
    Test<BACKEND>(d_keys_in,
                  d_values_in,
                  h_reference,
                  num_items,
                  scan_op,
                  initial_value,
                  equality_op);

    // Test in/out values aliasing
    TestInplaceValues<BACKEND>(d_keys_in,
                               d_values_in, 
                               h_reference,
                               num_items,
                               scan_op,
                               initial_value,
                               equality_op);

    CubDebugExit(hipMemcpy(d_values_in, h_values_in, sizeof(InputT) * num_items, hipMemcpyHostToDevice));

    // Test keys/values aliasing (should go last, changes keys)
    TestInplaceKeys<BACKEND>(d_keys_in,
                             d_values_in,
                             h_reference,
                             num_items,
                             scan_op,
                             initial_value,
                             equality_op);

    // Cleanup
    if (h_keys_in) delete[] h_keys_in;
    if (h_values_in) delete[] h_values_in;
    if (h_reference) delete[] h_reference;
    if (d_keys_in) CubDebugExit(g_allocator.DeviceFree(d_keys_in));
    if (d_values_in) CubDebugExit(g_allocator.DeviceFree(d_values_in));
}


/**
 * Test DeviceScan on iterator type
 */
template <
    Backend         BACKEND,
    typename        KeyT,
    typename        InputT,
    typename        OutputT,
    typename        ScanOpT,
    typename        InitialValueT,
    typename        EqualityOpT>
void TestIterator(
    int             num_items,
    ScanOpT         scan_op,
    InitialValueT   initial_value,
    EqualityOpT     equality_op)
{
    printf("\nIterator %s %s hipcub::DeviceScan::%s %d items, %s->%s (%d->%d bytes)\n",
        (BACKEND == CDP) ? "CDP CUB" : "CUB",
        (std::is_same<InitialValueT, NullType>::value) ? "Inclusive" : "Exclusive",
        (std::is_same<ScanOpT, Sum>::value) ? "Sum" : "Scan",
        num_items,
        typeid(InputT).name(), typeid(OutputT).name(), (int) sizeof(InputT), (int) sizeof(OutputT));
    fflush(stdout);

    // Use a counting iterator followed by div as the keys
    using CountingIterT = CountingInputIterator<int, int>;
    CountingIterT h_keys_in_helper(0);
    TransformInputIterator<KeyT, DivideByFiveFunctor<KeyT>, CountingIterT> h_keys_in(h_keys_in_helper, DivideByFiveFunctor<KeyT>());

    // Use a constant iterator as the input
    InputT val = InputT();
    ConstantInputIterator<InputT, int> h_values_in(val);

    // Allocate host arrays
    OutputT*  h_reference = new OutputT[num_items];

    // Initialize problem and solution
    Solve(h_keys_in, h_values_in, h_reference, num_items, scan_op, initial_value, equality_op);

    // Run Test
    Test<BACKEND>(h_keys_in, h_values_in, h_reference, num_items, scan_op, initial_value, equality_op);

    // Cleanup
    if (h_reference) delete[] h_reference;
}


/**
 * Test different gen modes
 */
template <
    Backend         BACKEND,
    typename        KeyT,
    typename        InputT,
    typename        OutputT,
    typename        ScanOpT,
    typename        InitialValueT,
    typename        EqualityOpT>
void Test(
    int             num_items,
    ScanOpT         scan_op,
    InitialValueT   initial_value,
    EqualityOpT     equality_op)
{
    TestPointer<BACKEND, KeyT, InputT, OutputT>(  num_items, UNIFORM, scan_op, initial_value, equality_op);
    TestPointer<BACKEND, KeyT, InputT, OutputT>(  num_items, RANDOM,  scan_op, initial_value, equality_op);
    TestIterator<BACKEND, KeyT, InputT, OutputT>( num_items, scan_op, initial_value, equality_op);
}


/**
 * Test different dispatch
 */
template <
    typename        KeyT,
    typename        InputT,
    typename        OutputT,
    typename        ScanOpT,
    typename        InitialValueT,
    typename        EqualityOpT>
void Test(
    int             num_items,
    ScanOpT         scan_op,
    InitialValueT   initial_value,
    EqualityOpT     equality_op)
{
    Test<CUB, KeyT, InputT, OutputT>(num_items, scan_op, initial_value, equality_op);
#ifdef CUB_CDP
    Test<CDP, KeyT, InputT, OutputT>(num_items, scan_op, initial_value, equality_op);
#endif
}


/**
 * Test different operators
 */
template <typename KeyT, typename InputT, typename OutputT, typename EqualityOpT>
void TestOp(
    int             num_items,
    OutputT         identity,
    OutputT         initial_value,
    EqualityOpT     equality_op)
{
    // Exclusive (use identity as initial value because it will dispatch to *Sum variants that don't take initial values)
    Test<KeyT, InputT, OutputT>(num_items, hipcub::Sum(), identity, equality_op);
    Test<KeyT, InputT, OutputT>(num_items, hipcub::Max(), identity, equality_op);

    // Exclusive (non-specialized, so we can test initial-value)
    Test<KeyT, InputT, OutputT>(num_items, WrapperFunctor<hipcub::Sum>(hipcub::Sum()), initial_value, equality_op);
    Test<KeyT, InputT, OutputT>(num_items, WrapperFunctor<hipcub::Max>(hipcub::Max()), initial_value, equality_op);

    // Inclusive (no initial value)
    Test<KeyT, InputT, OutputT>(num_items, hipcub::Sum(), NullType(), equality_op);
    Test<KeyT, InputT, OutputT>(num_items, hipcub::Max(), NullType(), equality_op);
}

/**
 * Test different key type and equality operator
 */
template <typename InputT, typename OutputT>
void TestKeyTAndEqualityOp(
    int             num_items,
    OutputT         identity,
    OutputT         initial_value)
{
    TestOp<bool, InputT>(num_items, identity, initial_value, Equality());
    TestOp<unsigned int, InputT>( num_items, identity, initial_value, Mod2Equality());
}

/**
 * Test different input sizes
 */
template <
    typename InputT,
    typename OutputT>
void TestSize(
    int         num_items,
    OutputT     identity,
    OutputT     initial_value)
{
    if (num_items < 0)
    {
        TestKeyTAndEqualityOp<InputT>(0,        identity, initial_value);
        TestKeyTAndEqualityOp<InputT>(1,        identity, initial_value);
        TestKeyTAndEqualityOp<InputT>(100,      identity, initial_value);
        TestKeyTAndEqualityOp<InputT>(10000,    identity, initial_value);
        TestKeyTAndEqualityOp<InputT>(1000000,  identity, initial_value);
    }
    else
    {
        TestKeyTAndEqualityOp<InputT>(num_items, identity, initial_value);
    }
}



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_items = -1;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_items);
    args.GetCmdLineArgument("i", g_timing_iterations);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<input items> "
            "[--i=<timing iterations> "
            "[--device=<device-id>] "
            "[--v] "
            "[--cdp]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());
    g_device_giga_bandwidth = args.device_giga_bandwidth;
    printf("\n");

    // %PARAM% TEST_VALUE_TYPES types 0:1:2:3:4:5

#if TEST_VALUE_TYPES == 0

    // Test different input+output data types
    TestSize<unsigned char>(num_items, (int)0, (int)99);

    // Test same input+output data types
    TestSize<unsigned char>(num_items, (unsigned char)0, (unsigned char)99);
    TestSize<char>(num_items, (char)0, (char)99);

#elif TEST_VALUE_TYPES == 1

    TestSize<unsigned short>(num_items, (unsigned short)0, (unsigned short)99);
    TestSize<unsigned int>(num_items, (unsigned int)0, (unsigned int)99);
    TestSize<unsigned long long>(num_items,
                                 (unsigned long long)0,
                                 (unsigned long long)99);
#elif TEST_VALUE_TYPES == 2

    TestSize<uchar2>(num_items, make_uchar2(0, 0), make_uchar2(17, 21));
    TestSize<char2>(num_items, make_char2(0, 0), make_char2(17, 21));
    TestSize<ushort2>(num_items, make_ushort2(0, 0), make_ushort2(17, 21));

#elif TEST_VALUE_TYPES == 3

    TestSize<uint2>(num_items, make_uint2(0, 0), make_uint2(17, 21));
    TestSize<ulonglong2>(num_items,
                         make_ulonglong2(0, 0),
                         make_ulonglong2(17, 21));
    TestSize<uchar4>(num_items,
                     make_uchar4(0, 0, 0, 0),
                     make_uchar4(17, 21, 32, 85));

#elif TEST_VALUE_TYPES == 4

    TestSize<char4>(num_items,
                    make_char4(0, 0, 0, 0),
                    make_char4(17, 21, 32, 85));

    TestSize<ushort4>(num_items,
                      make_ushort4(0, 0, 0, 0),
                      make_ushort4(17, 21, 32, 85));
    TestSize<uint4>(num_items,
                    make_uint4(0, 0, 0, 0),
                    make_uint4(17, 21, 32, 85));

#elif TEST_VALUE_TYPES == 5

    TestSize<ulonglong4>(num_items,
                         make_ulonglong4(0, 0, 0, 0),
                         make_ulonglong4(17, 21, 32, 85));

    TestSize<TestFoo>(num_items,
                      TestFoo::MakeTestFoo(0, 0, 0, 0),
                      TestFoo::MakeTestFoo(1ll << 63,
                                           1 << 31,
                                           static_cast<short>(1 << 15),
                                           static_cast<char>(1 << 7)));

    TestSize<TestBar>(num_items, TestBar(0, 0), TestBar(1ll << 63, 1 << 31));

#endif

    return 0;
}



